#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
	#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <windows.h>
#include < time.h >
//#include <iostream>

#include "scan.h"
#include "kernel.h"


//-------------------------------------------------------CPU TIMER LIBRARY-------------------------------------------------------

#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
#define DELTA_EPOCH_IN_MICROSECS  116444736000000000Ui64 // CORRECT
#else
#define DELTA_EPOCH_IN_MICROSECS  116444736000000000ULL // CORRECT
#endif

struct timezone
{
	int  tz_minuteswest; /* minutes W of Greenwich */
	int  tz_dsttime;     /* type of dst correction */
};

// Definition of a gettimeofday function
int gettimeofday(struct timeval* tv, struct timezone* tz)
{
	// Define a structure to receive the current Windows filetime
	FILETIME ft;

	// Initialize the present time to 0 and the timezone to UTC
	unsigned __int64 tmpres = 0;
	static int tzflag = 0;

	if (NULL != tv)
	{
		GetSystemTimeAsFileTime(&ft);

		// The GetSystemTimeAsFileTime returns the number of 100 nanosecond 
		// intervals since Jan 1, 1601 in a structure. Copy the high bits to 
		// the 64 bit tmpres, shift it left by 32 then or in the low 32 bits.
		tmpres |= ft.dwHighDateTime;
		tmpres <<= 32;
		tmpres |= ft.dwLowDateTime;

		// Convert to microseconds by dividing by 10
		tmpres /= 10;

		// The Unix epoch starts on Jan 1 1970.  Need to subtract the difference 
		// in seconds from Jan 1 1601.
		tmpres -= DELTA_EPOCH_IN_MICROSECS;

		// Finally change microseconds to seconds and place in the seconds value. 
		// The modulus picks up the microseconds.
		tv->tv_sec = (long)(tmpres / 1000000UL);
		tv->tv_usec = (long)(tmpres % 1000000UL);
	}

	if (NULL != tz)
	{
		if (!tzflag)
		{
			_tzset();
			tzflag++;
		}

		// Adjust for the timezone west of Greenwich
		tz->tz_minuteswest = _timezone / 60;
		tz->tz_dsttime = _daylight;
	}

	return 0;
}






//Takes arr[] as input and produces compact_0[] for bit-00, compact_1[] for bit-01, compact_2[] for bit-10 and compact_3[] for bit-11 , in bit-poisiton 'i' and 'i+1'
__global__ void compacter(unsigned int* arr, unsigned int* compact_0, unsigned int* compact_1, 
							unsigned int* compact_2, unsigned int* compact_3, int i)
{
	int idx = threadIdx.x;

	unsigned int num = arr[idx];

	//Predicate: Check if bit-0 is 1, then output is 1
	compact_0[idx] = (!((num >> (i + 1)) & 1)) && ( !((num >> i) & 1) ) ;    // '00' in bit-i+1 and bit-i
	compact_1[idx] =  ( !((num >> (i+1) ) & 1) ) && ( (num >> i ) & 1 ) ;    // '01' in bit-i+1 and bit-i
	compact_2[idx] =  ( (num >> (i + 1)) & 1) &&  ( ! ( (num >> i) & 1 ) )  ;    // '10' in bit-i+1 and bit-i
	compact_3[idx] =  ((num >> (i + 1)) & 1) && ((num >> i) & 1);				// '11' in bit-i+1 and bit-i
}


//Takes arr[], scatter addresses (scan of compact_0, 1, 2, 3, 4), compact_0,1,2,3[] as input, and produces sorted array for the respective bit, in arr[] itself
	//arr[] = Input  array to be sorted according to respective bit
	//scan_0[] = Scan array for compact of bit-0 in respective position
	//scan_1[] = Scan array for compact of bit-1 in respective position
	//compact_0[] = Compact array for bit-0 in respective position for arr[]
	//n = Size of arr[]
	//temp_out[] is temporary array used to store sorted array before copying back to arr[]
__global__ void scatter_sort(unsigned int* arr, unsigned int* scan_0, unsigned int* scan_1, unsigned int* scan_2, unsigned int* scan_3,
							 unsigned int* compact_0, unsigned int* compact_1, unsigned int* compact_2, unsigned int* compact_3,
							  int n, unsigned int* temp_out)
{
	int idx = threadIdx.x;

	int num[4];    // num[k]represents total number of elements with 'b1 b0' (binary of k), as bits at position i+1, i
	
	num[0] = scan_0[n];
	num[1] = scan_1[n];
	num[2] = scan_2[n];
	num[3] = scan_3[n];

	int pos;

	if (compact_0[idx] == 1)    //If element arr[idx] has bit-0 in current position
	{
		pos = scan_0[idx];    //The index position in output array where the arr[idx] is supposed to go
		temp_out[pos] = arr[idx];
	}

	else if (compact_1[idx] == 1)  //If element arr[idx] has bit-1 in current position
	{
		pos = scan_1[idx] + num[0];    //num_of_0 : Offset
		temp_out[pos] = arr[idx];
	}

	else if (compact_2[idx] == 1)  //If element arr[idx] has bit-1 in current position
	{
		pos = scan_2[idx] + num[0] + num[1];    //num_of_0 + num_of_1 : Offset
		temp_out[pos] = arr[idx];
	}

	else if (compact_3[idx] == 1)  //If element arr[idx] has bit-1 in current position
	{
		pos = scan_3[idx] + num[0] + num[1] + num[2];    //num_of_0 + num_of_1 + num_of_2 : Offset
		temp_out[pos] = arr[idx];
	}

	__syncthreads;   //Barrier


	//Copy from temp_out[] back to arr[]
	arr[idx] = temp_out[idx];

}


//Calculate scan for bit-1: 
	//scan_1[] is scan array for bit-1 that is to be calculated
	//scan_0[] is scan array for bit-0, given as input
	//n = size of array that is to be sorted
__global__ void scan_1_calculator(unsigned int* scan_1, unsigned int* scan_0, int n)
{
	int idx = threadIdx.x;
	int num_of_0_elements = scan_0[n];               //Number of elements with 0 in respective bit position
	int num_of_1_elements = n - num_of_0_elements;   //Number of elements with 1 in respective bit position = Total number of elements - Number of elements with 0 in respective bit position

	//Last element (at index n) of scan_1[] = num_of_1_elements 
	if ( idx == (n) )
	{
		scan_1[idx] = num_of_1_elements;
	}


	else
	{
		//Formula to calculate scan_1[i] using scan_1[i+1] and compact_1[]. 
		//Logic:
			//scan_1[i] = Total Number of 1-elements (elements with 1 at respective bit-position), till index i- in arr[]
		    //  i  -->index = Number of elements in arr[] before current element
			// scan_0[i]  --> Number of 0-elements till index-i in arr[]
		scan_1[idx] = idx - scan_0[idx];   
	}

}

void seq_bubble_sort(unsigned int* arr, int n)      //In-Place Sequential Bubble Sort
{
	int i, k, flag, temp;
	for (k = 1; k < (n - 1); k++)
	{
		flag = 0;
		for (i = 0; i < (n - k); i++)
		{
			if (arr[i] > arr[i + 1])
			{
				temp = arr[i];    //
				arr[i] = arr[i + 1];  //  Swapping A[i+1] and A[i]
				arr[i + 1] = temp;  //
			}
		}
	}
}

void radix_sort()
{

	//---------------------------------Create input arr[] and h_compact[] --------------------------------------------------------------------	

	//unsigned int h_arr[] = { 1989, 1124, 9701, 2900, 5241, 6702, 1784, 1096, 3382, 863, 8966, 2830, 2043, 9889, 3414, 2810, 644, 1420, 1065, 8597, 4419, 1388, 8796, 6139, 1158, 7689, 4114, 1865, 8485, 9190, 564, 5409, 9016, 3515, 2993, 6536, 7524, 2982, 9953, 3228, 1678, 2770, 7072, 3091, 9815, 7426, 363, 5139, 7481, 6183, 3392, 5808, 2407, 5425, 3353, 4484, 1825, 6621, 3046, 2710, 1533, 118, 7109, 2917, 4677, 641, 4802, 9366, 5029, 59, 9204, 9173, 5875, 7417, 9070, 2929, 1529, 6985, 5923, 9271, 1907, 8870, 7774, 3496, 2258, 5167, 7164, 9915, 2854, 1089, 1275, 1258, 7372, 6088, 375, 8333, 840, 3010, 5606, 2534, 138, 7266, 1339, 1714, 3615, 9601, 9817, 6208, 9483, 5269, 7835, 4681, 772, 7000, 6922, 6833, 4163, 6246, 3913, 5617, 7160, 2678, 8284, 8454, 7671, 8963, 4080, 4745, 6173, 8950, 4646, 1182, 4264, 3858, 2221, 270, 7341, 8382, 2889, 5722, 3947, 784, 4280, 439, 5331, 6080, 7296, 4075, 8380, 8049 };
	//unsigned int h_arr[] = { 26338, 1102, 43406, 62440, 7116, 75423, 95442, 31454, 43177, 27829, 37527, 20088, 48786, 33484, 91084, 98038, 93852, 64998, 84009, 91983, 12691, 8128, 90209, 64050, 64526, 63539, 31716, 52617, 55245, 69338, 96600, 62877, 9070, 89736, 64799, 45587, 22108, 78332, 86473, 6608, 39686, 67631, 90273, 41757, 57024, 65715, 6247, 35982, 49869, 83823, 50126, 83367, 58983, 78590, 50726, 61549, 96979, 9315, 99939, 92868, 42634, 72012, 75395, 66760, 97571, 67335, 50690, 55839, 7193, 68868, 52778, 50767, 38817, 73182, 73173, 18570, 59518, 75198, 43944, 81905, 17130, 30499, 62531, 67788, 10212, 93307, 33485, 35421, 41800, 91439, 7128, 5244, 4857, 19666, 55995, 93197, 16521, 70814, 65956, 29551, 41088, 90651, 8364, 81397, 65599, 21928, 79453, 43220, 30210, 89518, 46937, 39088, 3064, 8272, 95080, 72370, 38519, 80624, 73855, 14338, 26999, 797, 7389, 22675, 29441, 91215, 57751, 20691, 73766, 19449, 65664, 4911, 51829, 52694, 59465, 18815, 47909, 6960, 71232, 31887, 33836, 75839, 77099, 72696, 66743, 23003, 67416, 79256, 92027, 14854, 67547, 66948, 73557, 81593, 40373, 36769, 91653, 55675, 91273, 82840, 12390, 55618, 98120, 60072, 10352, 26671, 53834, 67112, 11591, 66281, 4508, 74787, 76274, 50136, 87370, 33150, 30572, 77426, 22738, 93988, 76158, 88211, 56058, 61264, 49590, 64482, 76270, 58166, 83076, 3572, 53961, 96103, 69219, 56519, 39247, 47900, 96419, 20127, 21434, 83026, 53972, 12275, 41420, 14470, 83492, 98815, 11219, 28147, 40874, 75086, 58471, 14116, 12447, 59360, 86793, 53957, 74870, 56028, 78651, 78761, 68152, 31585, 86419, 73739, 13907, 47057, 52096, 17777, 46550, 14465, 97001, 75372, 82870, 2510, 54067, 53592, 49157, 50366, 72633, 69869, 6182, 91356, 76672, 13479, 20844, 35748, 16099, 52512, 89438, 59887, 65339, 85400, 64340, 34066, 77991, 38365, 79942, 7545, 94624, 3357, 87348, 79218, 70170, 80566, 9464, 64659, 95658, 58290, 44600, 13764, 83274, 46833, 13879, 72497, 61917, 16609, 77559, 58483, 24873, 69414, 11725, 52316, 53275, 13306, 35063, 87894, 5532, 53202, 24268, 58043, 33283, 6492, 45431, 30406, 32716, 88618, 33055, 78184, 98307, 6890, 8994, 89487, 11234, 52622, 64764, 11958, 44815, 1532, 79736, 47961, 14542, 77658, 75199, 45576, 19542, 7035, 20812, 91288, 51135, 4249, 98847, 6941, 38347, 64487, 15925, 99375, 8921, 90455, 17026, 97407, 29745, 56171, 32655, 30296, 25287, 92752, 33062, 54188, 59874, 18971, 80723, 14297, 57746, 85341, 15519, 2459, 10977, 97487, 88749, 88970, 57457, 89542, 45360, 455, 65658, 10186, 96217, 87608, 45218, 90238, 76122, 54096, 77985, 97251, 8675, 9146, 38822, 80023, 87709, 7900, 80506, 24446, 51014, 14603, 94082, 43508, 43274, 67228, 56128, 37014, 43127, 43953, 96690, 35271, 67673, 71158, 50788, 69814, 76142, 791, 86797, 15482, 98654, 52733, 31128, 64146, 96229, 11191, 23915, 99656, 78797, 15635, 9645, 48483, 2935, 50237, 14411, 8588, 61111, 84174, 17764, 11618, 94105, 21335, 81908, 82632, 72680, 56202, 83381, 39008, 80899, 11145, 72512, 74094, 12690, 1350, 20388, 46408, 69910, 79823, 90323, 22544, 99050, 10961, 5284, 18371, 45704, 86015, 35965, 98364, 62855, 51699, 8593, 95676, 85731, 66224, 19520, 49572, 26118, 64236, 86854, 68097, 28618, 30063, 73433, 37923, 96022, 9800, 4261, 52449, 32161, 35946, 82185, 70005, 57610, 48663, 3915, 88572, 93349, 1709, 89097, 72236, 62070, 64663, 25783, 54343, 376, 57853, 6775, 81468, 32931, 44625, 91909, 52921, 84552, 94976, 65099, 37356, 86361, 34262, 571, 32338, 22506, 3641, 79365, 17411, 4160, 47729, 21059, 55742, 69135, 25617, 81408, 45984, 76631, 11955, 57728, 72173, 51879, 76821, 2146, 7880, 21508, 94189, 19944, 90717, 42518, 77249, 66967, 54725, 30968, 39205, 35117, 86598, 60500, 8489, 72419, 62481, 72656, 76243, 65476, 16709, 13908, 20416, 72724, 5089, 8265, 40097, 79298, 40171, 10218, 67885, 62477, 3068, 41719, 62229, 60152, 78163, 59021, 73703, 94992, 14615, 50868, 41622, 59259, 41659, 86480, 85254, 19229, 13745, 87784, 98089, 99842, 10269, 27592, 89481, 68467, 56126, 32884, 92184, 17521, 85446, 58156, 80504, 92966, 73456, 11176, 26435, 72496, 32129, 49957, 8609, 30759, 28815, 84018, 81335, 13435, 68014, 78931, 4503, 7175, 25168, 60915, 74655, 80498, 31761, 82006, 14226, 30640, 75321 };

	unsigned int h_arr[600] = { 3871, 3742, 1863, 2402, 904, 3951, 2762, 1795, 3884, 2894, 3505, 2386, 1871, 2802, 3890, 1051, 2738, 975, 1875, 1372, 107, 2, 1800, 1453, 742, 2564, 2012, 1332, 3359, 327, 291, 2288, 137, 3779, 1268, 2358, 3608, 134, 2031, 3770, 2817, 2307, 1463, 383, 1796, 3704, 2222, 2432, 2338, 416, 921, 3702, 114, 3375, 3191, 1854, 2810, 3331, 2087, 1763, 1842, 957, 3514, 2675, 1513, 202, 2436, 677, 2873, 2504, 2744, 103, 2964, 2997, 141, 285, 3333, 3497, 3512, 2861, 3278, 1384, 3009, 1874, 1536, 1062, 280, 365, 2119, 1809, 3669, 3963, 2274, 2334, 2773, 457, 1528, 1045, 3825, 1548, 223, 3063, 2724, 323, 1722, 1018, 1412, 3920, 3665, 2048, 616, 1245, 406, 124, 2231, 666, 840, 1519, 2572, 279, 2884, 316, 209, 1621, 3412, 963, 1315, 177, 980, 3475, 43, 1630, 1725, 680, 1627, 3497, 3552, 1401, 190, 102, 3254, 463, 2854, 663, 3984, 2604, 511, 2353, 65, 3900, 2476, 3144, 3997, 621, 633, 2867, 2250, 2958, 3031, 3063, 3605, 863, 752, 1443, 459, 2689, 3500, 278, 643, 56, 1934, 1140, 3868, 2692, 3287, 3134, 3575, 1961, 2400, 1960, 3054, 3911, 3811, 1242, 3866, 3677, 501, 725, 2042, 1306, 957, 1312, 2180, 3708, 1932, 2619, 3720, 3681, 2887, 153, 513, 360, 2032, 2050, 1086, 3553, 3594, 213, 950, 21, 1435, 610, 1369, 877, 367, 2557, 3444, 98, 3635, 1839, 2828, 2167, 3989, 3850, 3549, 1882, 2498, 2263, 2718, 3522, 1180, 3080, 3004, 3645, 1582, 2193, 3087, 3977, 2647, 10, 2440, 1054, 1785, 1441, 1383, 2020, 616, 3122, 2255, 1362, 3668, 3369, 1419, 69, 1843, 2458, 125, 2226, 1856, 2916, 428, 1703, 1279, 2236, 2016, 2963, 791, 165, 2535, 2819, 3806, 2728, 2820, 2929, 3665, 1422, 2733, 1430, 3988, 1571, 2716, 1236, 1689, 2262, 2746, 2162, 2322, 951, 2029, 2890, 3607, 2771, 2867, 2015, 3205, 1011, 3197, 3367, 1089, 1776, 1537, 1611, 298, 1067, 2537, 86, 549, 2708, 3010, 407, 3120, 2722, 1028, 239, 1881, 805, 1612, 2568, 789, 3755, 1045, 3483, 1495, 927, 660, 262, 28, 163, 2483, 2423, 3596, 2391, 3082, 220, 3378, 2752, 3209, 1616, 3405, 2639, 653, 3237, 3015, 3417, 3875, 961, 3932, 759, 118, 1441, 1001, 760, 3980, 1114, 981, 2681, 1182, 3707, 2761, 881, 2693, 323, 936, 2473, 1046, 1889, 3058, 2468, 2381, 2609, 910, 1847, 318, 2155, 3776, 1705, 3334, 2009, 1538, 2966, 2929, 435, 197, 21, 3489, 3751, 2209, 2426, 2329, 835, 3059, 412, 92, 383, 1317, 795, 3456, 2615, 2760, 3586, 3129, 1183, 3369, 1940, 316, 3004, 748, 3793, 3917, 1241, 3145, 1540, 1591, 3403, 3771, 627, 3213, 2179, 2199, 957, 745, 364, 3649, 1294, 101, 1027, 2682, 3508, 3262, 2357, 2062, 1704, 370, 841, 3553, 1029, 2438, 620, 320, 3368, 3810, 3373, 2617, 3072, 1057, 2023, 2138, 2712, 3769, 456, 3717, 321, 2757, 1618, 387, 1083, 2582, 1073, 2509, 2856, 3041, 2374, 2319, 1225, 125, 1783, 195, 1203, 3195, 2142, 3441, 2096, 113, 2113, 3151, 815, 3224, 2175, 2590, 2986, 3608, 486, 720, 2276, 553, 1279, 650, 825, 988, 2737, 310, 1944, 1225, 1205, 681, 3886, 3097, 3350, 79, 1900, 1863, 2072, 1851, 2926, 2507, 3396, 2226, 1035, 377, 3929, 3050, 3643, 51, 827, 1652, 3409, 1942, 2810, 2639, 1955, 2317, 1068, 1847, 1240, 1275, 2173, 2844, 1174, 3445, 3502, 482, 2693, 353, 2840, 3291, 548, 3480, 1767, 648, 1745, 2854, 8, 3780, 2827, 1243, 1917, 1184, 1559, 1524, 673, 1723, 3387, 1583, 2526, 2677, 2364, 1933, 3310, 2922, 3932, 2571, 98, 1228, 2162, 108, 974, 3168, 3369, 2674, 1049, 3714, 2725, 2340, 2224, 3641, 240, 1073, 469, 1580, 1385, 931, 963, 3474, 183, 2598, 1804, 373, 2318, 1195, 1308, 3331, 3946, 2104, 3146, 64, 901, 3478, 1527, 3516, 1525 };
	int n = sizeof(h_arr) / sizeof(unsigned int);   //Size of input array
	unsigned int* h_compact = new unsigned int[n];


	//------------------Create d_arr[], d_compact_0[], d_compact_1[], d_compact_2[], d_compact_3[] d_temp_out[]------------------------------------------ 
	unsigned int* d_arr, *d_compact_0, *d_compact_1, * d_compact_2, * d_compact_3, *d_temp_out;
	//d_temp_out[] is temporary array used to temporarily store sorted output in kernel

	hipMalloc((void**) &d_arr, n * sizeof(unsigned int));   //Allocate d_arr[] in GPU for n elements
	hipMemcpy((void*) d_arr, (void*) h_arr, n * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMalloc((void**) &d_compact_0, n * sizeof(unsigned int));   //Allocate compact_0 array [] in GPU for n elements
	hipMalloc((void**) &d_compact_1, n * sizeof(unsigned int));
	hipMalloc((void**) &d_compact_2, n * sizeof(unsigned int));
	hipMalloc((void**) &d_compact_3, n * sizeof(unsigned int));
	hipMalloc((void**) &d_temp_out, n * sizeof(unsigned int));   //Allocate compact_1 array [] in GPU for n elements



	//-------------------------------------------------Create Scan Arrays in Host--------------------------------------------------
		//n+1 because we need 1 extra element to hold scan of arr[i+1]
	unsigned int* h_scan_0 = new unsigned int [n+1];
	unsigned int* h_scan_1 = new unsigned int [n+1];
	unsigned int* h_scan_2 = new unsigned int [n+1];
	unsigned int* h_scan_3 = new unsigned int [n+1];
	

	//-------------------------------------------------Create Scan Arrays in Device--------------------------------------------------
	unsigned int* d_scan_0;   //For bit-00
	hipMalloc((void**) &d_scan_0, (n+1) * sizeof(unsigned int));   //Allocate compact_0 array [] in GPU for n+1 elements

	unsigned int* d_scan_1;   //For bit-01
	hipMalloc((void**) &d_scan_1, (n+1) * sizeof(unsigned int));

	unsigned int* d_scan_2;   //For bit-10
	hipMalloc((void**)&d_scan_2, (n + 1) * sizeof(unsigned int));

	unsigned int* d_scan_3;   //For bit-11
	hipMalloc((void**)&d_scan_3, (n + 1) * sizeof(unsigned int));

	GpuTimer timer;
	timer.Start();

	// ---------------------------Iterate 32 times (1 time for each bit of integer) and Make Kernel Call-------------------------
	int bit_pair;   //First index of bit-pair in arr[]
	for (bit_pair = 0; bit_pair <= 30; bit_pair = bit_pair+2)
	{
		//int bit_pair = 0;
		compacter <<< 1, n >>> (d_arr, d_compact_0, d_compact_1, d_compact_2, d_compact_3, bit_pair);     //Launch kernel : 1 block of n threads

		//Produce scan arrays
		sum_scan_blelloch(d_scan_0, d_compact_0, n + 1);
		sum_scan_blelloch(d_scan_1, d_compact_1, n + 1);
		sum_scan_blelloch(d_scan_2, d_compact_2, n + 1);
		sum_scan_blelloch(d_scan_3, d_compact_3, n + 1);

		scatter_sort <<<1, n>>>(d_arr, d_scan_0, d_scan_1, d_scan_2, d_scan_3, d_compact_0, d_compact_1, d_compact_2, d_compact_3, n, d_temp_out);

	}
	
	timer.Stop();
	double time_elapsed = timer.Elapsed();

	/*
	//--------------------------------Copy output to CPU h_out[] and print sorted array-----------------------------------------------------------
	unsigned int* h_out = new unsigned int[n];
	hipMemcpy((void*) h_out, (void*) d_arr, (n) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++)
	{
		printf("%d ", h_out[i]);
	}
	*/


	
	hipMemcpy((void*)h_scan_0, (void*)d_arr, (n) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	/*
	hipMemcpy((void*)h_scan_1, (void*)d_scan_1, (n+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy((void*)h_scan_2, (void*)d_scan_2, (n+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy((void*)h_scan_3, (void*)d_scan_3, (n+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	*/


	for (int i = 0; i < n; i++)
	{
		//printf("%d		%d		%d		%d \n", h_scan_0[i], h_scan_1[i], h_scan_2[i], h_scan_3[i]);
		printf("%d ", h_scan_0[i]);
	}

	printf("\n Time Elapsed : %g ms", time_elapsed);
	/*
	//Copy output
	hipMemcpy((void*)h_scan_0, (void*)d_scan_0, (n+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy((void*)h_scan_1, (void*)d_scan_1, (n+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);


	for (int i = 0; i < 9; i++)
	{
		printf("%d ", h_scan_0[i]);
	}
	printf("\n");

	for (int i = 0; i < 9; i++)
	{
		printf("%d ", h_scan_1[i]);
	}
	*/
}


void main()
{
	

	radix_sort();

	/*
	struct timeval timediff;

	
	gettimeofday(&timediff, NULL);
	double t1 = timediff.tv_sec + (timediff.tv_usec / 1000000.0);

	unsigned int h_arr[] = { 3871, 3742, 1863, 2402, 904, 3951, 2762, 1795, 3884, 2894, 3505, 2386, 1871, 2802, 3890, 1051, 2738, 975, 1875, 1372, 107, 2, 1800, 1453, 742, 2564, 2012, 1332, 3359, 327, 291, 2288, 137, 3779, 1268, 2358, 3608, 134, 2031, 3770, 2817, 2307, 1463, 383, 1796, 3704, 2222, 2432, 2338, 416, 921, 3702, 114, 3375, 3191, 1854, 2810, 3331, 2087, 1763, 1842, 957, 3514, 2675, 1513, 202, 2436, 677, 2873, 2504, 2744, 103, 2964, 2997, 141, 285, 3333, 3497, 3512, 2861, 3278, 1384, 3009, 1874, 1536, 1062, 280, 365, 2119, 1809, 3669, 3963, 2274, 2334, 2773, 457, 1528, 1045, 3825, 1548, 223, 3063, 2724, 323, 1722, 1018, 1412, 3920, 3665, 2048, 616, 1245, 406, 124, 2231, 666, 840, 1519, 2572, 279, 2884, 316, 209, 1621, 3412, 963, 1315, 177, 980, 3475, 43, 1630, 1725, 680, 1627, 3497, 3552, 1401, 190, 102, 3254, 463, 2854, 663, 3984, 2604, 511, 2353, 65, 3900, 2476, 3144, 3997, 621, 633, 2867, 2250, 2958, 3031, 3063, 3605, 863, 752, 1443, 459, 2689, 3500, 278, 643, 56, 1934, 1140, 3868, 2692, 3287, 3134, 3575, 1961, 2400, 1960, 3054, 3911, 3811, 1242, 3866, 3677, 501, 725, 2042, 1306, 957, 1312, 2180, 3708, 1932, 2619, 3720, 3681, 2887, 153, 513, 360, 2032, 2050, 1086, 3553, 3594, 213, 950, 21, 1435, 610, 1369, 877, 367, 2557, 3444, 98, 3635, 1839, 2828, 2167, 3989, 3850, 3549, 1882, 2498, 2263, 2718, 3522, 1180, 3080, 3004, 3645, 1582, 2193, 3087, 3977, 2647, 10, 2440, 1054, 1785, 1441, 1383, 2020, 616, 3122, 2255, 1362, 3668, 3369, 1419, 69, 1843, 2458, 125, 2226, 1856, 2916, 428, 1703, 1279, 2236, 2016, 2963, 791, 165, 2535, 2819, 3806, 2728, 2820, 2929, 3665, 1422, 2733, 1430, 3988, 1571, 2716, 1236, 1689, 2262, 2746, 2162, 2322, 951, 2029, 2890, 3607, 2771, 2867, 2015, 3205, 1011, 3197, 3367, 1089, 1776, 1537, 1611, 298, 1067, 2537, 86, 549, 2708, 3010, 407, 3120, 2722, 1028, 239, 1881, 805, 1612, 2568, 789, 3755, 1045, 3483, 1495, 927, 660, 262, 28, 163, 2483, 2423, 3596, 2391, 3082, 220, 3378, 2752, 3209, 1616, 3405, 2639, 653, 3237, 3015, 3417, 3875, 961, 3932, 759, 118, 1441, 1001, 760, 3980, 1114, 981, 2681, 1182, 3707, 2761, 881, 2693, 323, 936, 2473, 1046, 1889, 3058, 2468, 2381, 2609, 910, 1847, 318, 2155, 3776, 1705, 3334, 2009, 1538, 2966, 2929, 435, 197, 21, 3489, 3751, 2209, 2426, 2329, 835, 3059, 412, 92, 383, 1317, 795, 3456, 2615, 2760, 3586, 3129, 1183, 3369, 1940, 316, 3004, 748, 3793, 3917, 1241, 3145, 1540, 1591, 3403, 3771, 627, 3213, 2179, 2199, 957, 745, 364, 3649, 1294, 101, 1027, 2682, 3508, 3262, 2357, 2062, 1704, 370, 841, 3553, 1029, 2438, 620, 320, 3368, 3810, 3373, 2617, 3072, 1057, 2023, 2138, 2712, 3769, 456, 3717, 321, 2757, 1618, 387, 1083, 2582, 1073, 2509, 2856, 3041, 2374, 2319, 1225, 125, 1783, 195, 1203, 3195, 2142, 3441, 2096, 113, 2113, 3151, 815, 3224, 2175, 2590, 2986, 3608, 486, 720, 2276, 553, 1279, 650, 825, 988, 2737, 310, 1944, 1225, 1205, 681, 3886, 3097, 3350, 79, 1900, 1863, 2072, 1851, 2926, 2507, 3396, 2226, 1035, 377, 3929, 3050, 3643, 51, 827, 1652, 3409, 1942, 2810, 2639, 1955, 2317, 1068, 1847, 1240, 1275, 2173, 2844, 1174, 3445, 3502, 482, 2693, 353, 2840, 3291, 548, 3480, 1767, 648, 1745, 2854, 8, 3780, 2827, 1243, 1917, 1184, 1559, 1524, 673, 1723, 3387, 1583, 2526, 2677, 2364, 1933, 3310, 2922, 3932, 2571, 98, 1228, 2162, 108, 974, 3168, 3369, 2674, 1049, 3714, 2725, 2340, 2224, 3641, 240, 1073, 469, 1580, 1385, 931, 963, 3474, 183, 2598, 1804, 373, 2318, 1195, 1308, 3331, 3946, 2104, 3146, 64, 901, 3478, 1527, 3516, 1525 };
	int n = sizeof(h_arr) / sizeof(unsigned int);   //Size of input array

	//seq_bubble_sort(h_arr, n);

	
	gettimeofday(&timediff, NULL);
	double t2 = timediff.tv_sec + (timediff.tv_usec / 1000000.0);
	*/

}

